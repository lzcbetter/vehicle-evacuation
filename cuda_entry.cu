#include "hip/hip_runtime.h"
/*
 *  file name: cuda_entry.cu
 *
 * 
 * nvcc -c cuda_entry.c -lcurand
 * or compile against the static cuRAND library
 * nvcc -c cuda_entry.c -lcurand_static -lculibos
 */
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <vector_functions.h>
#include <cstdlib>
#include <fstream>

#define CUDA_BLOCK_SIZE    32
#define VEHICLE_PER_STEP   1.5
#define EPS                1e-5
#define ENV_DIM_X          200
#define ENV_DIM_Y          200
#define N_ITER             500


using namespace std;
/*
***********************************************************************************************************
* 
*                    Global Variables
*                   
***********************************************************************************************************
*/
// hiprand library uses hiprandState_t to keep track of the seed value 
// we will store a random state for every thread 
hiprandState_t* curand_states;
// duplicate the last one to avoid random number generater hiprand_uniform generate exactly 1.0
__constant__  unsigned char order[25][4] = {{0,1,2,3}, {0,1,3,2}, {0,2,1,3}, {0,2,3,1}, 
                                            {0,3,1,2}, {0,3,2,1}, {1,0,2,3}, {1,0,3,2}, 
                                            {1,2,0,3}, {1,2,3,0}, {1,3,0,2}, {1,3,2,0}, 
                                            {2,0,1,3}, {2,0,3,1}, {2,1,0,3}, {2,1,3,0}, 
                                            {2,3,0,1}, {2,3,1,0}, {3,0,1,2}, {3,0,2,1}, 
                                            {3,1,0,2}, {3,1,2,0}, {3,2,0,1}, {3,2,1,0}, {3,2,1,0}};
/*
***********************************************************************************************************
* func   name: curand_init_all
* description: this GPU kernel function is used to initialize the random states
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
__global__ void curand_init_all(unsigned int seed, hiprandState_t* states, int Ngx, int Ngy) {
    int g_idx = blockIdx.x*blockDim.x + threadIdx.x;
    int g_idy = blockIdx.y*blockDim.y + threadIdx.y;
    int uni_id = g_idy * Ngx + g_idx;
    if(g_idx >= Ngx || g_idy >= Ngy)
    {
        return;
    }
    hiprand_init(seed,       /* the seed can be the same for each core, here we pass the time in from the CPU */
                uni_id,     /* the sequence number should be different for each core (unless you want all
                               cores to get the same sequence of numbers for some reason - use thread id! */
                0,          /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[uni_id]);
}

/*
***********************************************************************************************************
* func   name: evacuation_update
* description: this GPU kernel function is used to foward the simulation one step. each thread will process
               one cell, map was divided to cell as an intersection model (corresponding turn probabilities 
               were set as zero if the cell is not a real intersection in reality).
* parameters :
*             none
* return: none
* note:   cuda vec 4 type: x->north; y->east; z->south; w->west; 
***********************************************************************************************************
*/
__global__ void evacuation_update(float *cnt, float *cap, float4 *pturn, 
                                  int Ngx, int Ngy, float * d_halo_sync, hiprandState_t* states) 
{
    int g_idx = blockIdx.x*blockDim.x + threadIdx.x;
    int g_idy = blockIdx.y*blockDim.y + threadIdx.y;
    int uni_id = g_idy * Ngx + g_idx;
    
    if(g_idx >= Ngx || g_idy >= Ngy)
    {
        return;
    }
    __shared__ float4 io[CUDA_BLOCK_SIZE+2][CUDA_BLOCK_SIZE+2];
    __shared__ float halo_sync[4][CUDA_BLOCK_SIZE];  // order: N -> E -> S -> W
    
    float cnt_temp = cnt[uni_id];
    int idx = threadIdx.x + 1, idy = threadIdx.y + 1;
// 1st step, fill in current [r, c] with # of outing vehicle, (determine # of care go L/R/U/S)
// note that, this is NOT the number of vehicles will be out after the current step
// it depends on the saturation of neighboors, but sure will not be more than the outgoing capacity(depends on speed of vehicle)
    float cnt_out = fminf(VEHICLE_PER_STEP, cnt_temp);
    float4 pturn_c = pturn[uni_id];          // turn probabilities of the cell [i, j]
    io[idy][idx].x = cnt_out * pturn_c.x;    // go north
    io[idy][idx].y = cnt_out * pturn_c.y;    // go east
    io[idy][idx].z = cnt_out * pturn_c.z;    // go south
    io[idy][idx].w = cnt_out * pturn_c.w;    // go west

    // extra work for edge threads, for the halo
    if(idx == 0){
        pturn_c = pturn[uni_id-1];
        cnt_out = fminf(VEHICLE_PER_STEP, cnt[uni_id-1]);
        io[idy][0].x = cnt_out * pturn_c.x;    // go north
        io[idy][0].y = cnt_out * pturn_c.y;    // go east
        io[idy][0].z = cnt_out * pturn_c.z;    // go south
        io[idy][0].w = cnt_out * pturn_c.w;    // go west    
        halo_sync[3][idy] = io[idy][0].y;      // will be used to computing how many vehicles get accepted by west cell
    }
    if(idx == CUDA_BLOCK_SIZE-1){
        pturn_c = pturn[uni_id+1];
        cnt_out = fminf(VEHICLE_PER_STEP, cnt[uni_id+1]);
        io[idy][CUDA_BLOCK_SIZE+1].x = cnt_out * pturn_c.x;    // go north
        io[idy][CUDA_BLOCK_SIZE+1].y = cnt_out * pturn_c.y;    // go east
        io[idy][CUDA_BLOCK_SIZE+1].z = cnt_out * pturn_c.z;    // go south
        io[idy][CUDA_BLOCK_SIZE+1].w = cnt_out * pturn_c.w;    // go west   
        halo_sync[1][idy] = io[idy][CUDA_BLOCK_SIZE+1].w;
    }

    if(idy == 0){
        pturn_c = pturn[uni_id-Ngx];
        cnt_out = fminf(VEHICLE_PER_STEP, cnt[uni_id-Ngx]);
        io[0][idx].x = cnt_out * pturn_c.x;    // go north
        io[0][idx].y = cnt_out * pturn_c.y;    // go east
        io[0][idx].z = cnt_out * pturn_c.z;    // go south
        io[0][idx].w = cnt_out * pturn_c.w;    // go west          
        halo_sync[0][idx] = io[0][idx].z;
    }    
    if(idy == CUDA_BLOCK_SIZE-1){
        pturn_c = pturn[uni_id+Ngx];
        cnt_out = fminf(VEHICLE_PER_STEP, cnt[uni_id+Ngx]);        
        io[CUDA_BLOCK_SIZE+1][idx].x = cnt_out * pturn_c.x;    // go north
        io[CUDA_BLOCK_SIZE+1][idx].y = cnt_out * pturn_c.y;    // go east
        io[CUDA_BLOCK_SIZE+1][idx].z = cnt_out * pturn_c.z;    // go south
        io[CUDA_BLOCK_SIZE+1][idx].w = cnt_out * pturn_c.w;    // go west    
        halo_sync[2][idx] = io[CUDA_BLOCK_SIZE+1][idx].x;      
    }
    // then wait untill all the threads in the sam thread block finish their outgoing conut processing
    __syncthreads();  
// 2nd step, process incoming vehicles, it will update outgoing requests of neighboors. 
    float diff_cap = cap[uni_id] - cnt_temp;                   // the capacity of incoming vehicles 
    float diff_bk = diff_cap;                                  // save the capacity for computing how many vehicles entered at the end
    /// priority ? random
    // returns a random number between 0.0 and 1.0 following a uniform distribution.
    int rnd = (unsigned char)( hiprand_uniform(&states[uni_id])*24 ); 
    for (int i=0; i<4 && diff_cap > EPS; i++)
    {
        switch(order[rnd][i])
        {
            case 0:
                if(diff_cap > io[idx][idy-1].z)
                {
                    diff_cap -= io[idx][idy-1].z;
                    io[idx][idy-1].z = 0.f;
                }else{
                    io[idx][idy-1].z -= diff_cap;
                    diff_cap = 0.0;
                }
                break;
            case 1:
                if(diff_cap > io[idx+1][idy].w)
                {
                    diff_cap -= io[idx+1][idy].w;
                    io[idx+1][idy].w = 0.f;
                }else{
                    io[idx+1][idy].w -= diff_cap;
                    diff_cap = 0.0;
                }
                break;
            case 2:
                if(diff_cap > io[idx][idy+1].x)
                {
                    diff_cap -= io[idx][idy+1].x;
                    io[idx][idy+1].x = 0.f;
                }else{
                    io[idx][idy+1].x -= diff_cap;
                    diff_cap = 0.0;
                }
                break;
            case 3:
                if(diff_cap > io[idx-1][idy].y)
                {
                    diff_cap -= io[idx-1][idy].y;
                    io[idx-1][idy].y = 0.f;
                }else{
                    io[idx-1][idy].y -= diff_cap;
                    diff_cap = 0.0;
                }
                break;                                                            
        }
    } 

    __syncthreads();
// add saturated vehicle back to counter, pre_cnt - (want_go - saturated) + incoming(in_cap - in_cap_left)
    cnt[uni_id] = cnt_temp - (cnt_out - io[idy][idx].x - io[idy][idx].y - io[idy][idx].z - io[idy][idx].w) 
                + (diff_bk - diff_cap);
   __syncthreads();
   
// 3rd step, process halo synchronization!!!! synchronizing via device global memory
    // to update, we have to know how much vehicle actully went out (get accepted by neighboor)
    int blk_uid = blockIdx.y*gridDim.x + blockIdx.x;
    int id_helper = blk_uid * (4 * CUDA_BLOCK_SIZE);
    if(idx == 0){                                // left
        id_helper += 3*CUDA_BLOCK_SIZE + threadIdx.y;
        d_halo_sync[id_helper] = halo_sync[3][idy] - io[idy][0].y;   // number of vehicles which actully go out
    }      
    if(idx == CUDA_BLOCK_SIZE-1){                // right
        id_helper += CUDA_BLOCK_SIZE + threadIdx.y;
        d_halo_sync[id_helper] = halo_sync[1][idy] - io[idy][CUDA_BLOCK_SIZE+1].w;
    }

    if(idy == 0){                                // top
        id_helper += threadIdx.x;
        d_halo_sync[id_helper] = halo_sync[0][idx] - io[0][idx].z;
    }

    if(idy == CUDA_BLOCK_SIZE-1){                // bottom
        id_helper += 2*CUDA_BLOCK_SIZE + threadIdx.x;
        d_halo_sync[id_helper] = halo_sync[2][idx] - io[CUDA_BLOCK_SIZE+1][idx].x;
    }       
}

/*
***********************************************************************************************************
* func   name: evacuation_halo_sync
* description: this GPU kernel function is used to sync cuda block edge.
* parameters :
*             none
* return: none
* note:   cuda vec 4 type: x->north; y->east; z->south; w->west; 
***********************************************************************************************************
*/
__global__ void evacuation_halo_sync(float *cnt, float *cap, float4 *pturn, 
                                     int Ngx, int Ngy, float * d_halo_sync) 
{
    int g_idx = blockIdx.x*blockDim.x + threadIdx.x;
    int g_idy = blockIdx.y*blockDim.y + threadIdx.y;
    int uni_id = g_idy * Ngx + g_idx;   
    if(g_idx >= Ngx || g_idy >= Ngy)
    {
        return;
    }    
	int idx = threadIdx.x + 1, idy = threadIdx.y + 1; 
    if(idx == 0 && blockIdx.x > 0){                                  // left
        int id_helper = (blockIdx.y*gridDim.x + blockIdx.x - 1) * (4 * CUDA_BLOCK_SIZE);
        id_helper += 3*CUDA_BLOCK_SIZE + threadIdx.y;
        cnt[uni_id] -= d_halo_sync[id_helper];  
    }      
    if(idx == CUDA_BLOCK_SIZE-1 && blockIdx.x < gridDim.x-1){        // right
        int id_helper = (blockIdx.y*gridDim.x + blockIdx.x + 1) * (4 * CUDA_BLOCK_SIZE);
        id_helper += CUDA_BLOCK_SIZE + threadIdx.y;
        cnt[uni_id] -= d_halo_sync[id_helper]; 
    }

    if(idy == 0 && blockIdx.y > 0){                                  // top
        int id_helper = ( (blockIdx.y-1)*gridDim.x + blockIdx.x) * (4 * CUDA_BLOCK_SIZE);
        id_helper += threadIdx.x;
        cnt[uni_id] -= d_halo_sync[id_helper]; 
    }

    if(idy == CUDA_BLOCK_SIZE-1 && blockIdx.y < gridDim.y-1){        // bottom
        int id_helper = ( (blockIdx.y+1)*gridDim.x + blockIdx.x) * (4 * CUDA_BLOCK_SIZE);
        id_helper += 2*CUDA_BLOCK_SIZE + threadIdx.x;
        cnt[uni_id] -= d_halo_sync[id_helper]; 
    }       
    
}
/*
***********************************************************************************************************
* func   name: evacuation_cuda_init
* description: initialize cuda related variable/environment, this function should be called in 
               model initialization
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
void evacuation_cuda_init(int Ngx, int Ngy){
    int nthread = Ngx * Ngy;
    // allocate space on the GPU for the random states 
    hipMalloc((void**) &curand_states, nthread * sizeof(hiprandState_t));
    
    // Launch configuration:
    dim3 dimBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE, 1);
    dim3 dimGrid(ceil((float)Ngx/CUDA_BLOCK_SIZE), ceil((float)Ngy/CUDA_BLOCK_SIZE), 1);
    
    // invoke the GPU to initialize all of the random states 
    curand_init_all<<<dimGrid, dimBlock>>>(time(0), curand_states, Ngx, Ngy);
}
/*
***********************************************************************************************************
* func   name: evacuation_3D_gpu_finalize
* description: release allocated resource in cuda runtime, this function should be called 
               in model finalize func.
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
void evacuation_cuda_finalize()
{
    hipFree(curand_states);
}

/*
***********************************************************************************************************
* func   name: evacuation_field_init
* description: initialize the field, i.e., initialize all the turn probabilities 
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
void evacuation_field_init(float4 *p_turn)
{
    for(int r = 0; r < ENV_DIM_Y; r++){
        for(int c = 0; c < ENV_DIM_X; c++){
            int idx = r*ENV_DIM_X+c;
            p_turn[idx].x = 0.1;
            p_turn[idx].y = 0.7;
            p_turn[idx].z = 0.1;
            p_turn[idx].w = 0.1;
        }
    }
}

/*
***********************************************************************************************************
* func   name: evacuation_state_init
* description: initialize the state, i.e., initialize number of vehicles in each of the cells, and capacity 
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
void evacuation_state_init(float *p_cnt, float *p_cap)
{
    for(int r = 0; r < ENV_DIM_Y; r++){
        for(int c = 0; c < ENV_DIM_X; c++){
            int idx = r*ENV_DIM_X+c;
            p_cap[idx] = 10.0;
            p_cnt[idx] = p_cap[idx] * rand() / RAND_MAX;
        }
    }
}
/*
***********************************************************************************************************
* func   name: write_vehicle_cnt_info
* description: write results to file for visualizing
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
void write_vehicle_cnt_info(int time_step, float * p_vcnt)
{
    ofstream output_file;
    char filename[100];
    sprintf( filename, "vehicle-cnt-info-ts%d.txt", time_step);
    output_file.open(filename);
    for(int r = 0; r < ENV_DIM_Y; r++){
        for(int c = 0; c < ENV_DIM_X; c++){
            int idx = r*ENV_DIM_X+c;
            output_file << p_vcnt[idx] << ",";
        }
        output_file << endl;
    }    
    output_file.close();
}
/*
***********************************************************************************************************
* func   name: main
* description: main entry of the model implementation
* parameters :
*             none
* return: none
***********************************************************************************************************
*/
int main()
{
    int Ngx = ENV_DIM_X, Ngy = ENV_DIM_Y;
    // this device memory is used for sync block halo, i.e., halo evacuation
    float *d_helper;                             // order: north -> east -> south -> west
    hipError_t cuda_error;
    float *h_vcnt = new float[ENV_DIM_X*ENV_DIM_Y];
    float *h_vcap = new float[ENV_DIM_X*ENV_DIM_Y];
    float4 *h_turn = new float4[ENV_DIM_X*ENV_DIM_Y];
    evacuation_field_init(h_turn);
    evacuation_state_init(h_vcnt, h_vcap);
    float *d_vcnt, *d_vcap;
    float4 *d_turn;
    cuda_error = hipMalloc((void**)&d_vcnt, sizeof(float)*ENV_DIM_X*ENV_DIM_Y);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMalloc: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }
    cuda_error = hipMalloc((void**)&d_vcap, sizeof(float)*ENV_DIM_X*ENV_DIM_Y);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMalloc: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }
    cuda_error = hipMalloc((void**)&d_turn, sizeof(float4)*ENV_DIM_X*ENV_DIM_Y);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMalloc: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }     
    // copy data from host to device
    cuda_error = hipMemcpy((void *)d_vcnt, (void *)h_vcnt, sizeof(float)*ENV_DIM_X*ENV_DIM_Y, hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMemcpy: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }  
    cuda_error = hipMemcpy((void *)d_vcap, (void *)h_vcap, sizeof(float)*ENV_DIM_X*ENV_DIM_Y, hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMemcpy: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }  
    cuda_error = hipMemcpy((void *)d_turn, (void *)h_turn, sizeof(float4)*ENV_DIM_X*ENV_DIM_Y, hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMemcpy: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }    
       
    dim3 dimBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE, 1);
    dim3 dimGrid(ceil((float)Ngx/CUDA_BLOCK_SIZE), ceil((float)Ngy/CUDA_BLOCK_SIZE), 1);
    int helper_size = 4 * CUDA_BLOCK_SIZE * dimGrid.x * dimGrid.y;
    cuda_error = hipMalloc((void**)&d_helper, helper_size);
    if (cuda_error != hipSuccess)
    {
        cout << "CUDA error in hipMalloc: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(evacuation_update), hipFuncCachePreferShared);
    
    for(int i = 0; i < N_ITER; i++){
        evacuation_update<<<dimGrid, dimBlock>>>(d_vcnt, d_vcap, d_turn, ENV_DIM_X, ENV_DIM_Y, d_helper, curand_states);
        hipDeviceSynchronize();
        evacuation_halo_sync<<<dimGrid, dimBlock>>>(d_vcnt, d_vcap, d_turn, ENV_DIM_X, ENV_DIM_Y, d_helper);
        hipDeviceSynchronize();
        if(i%50 == 0) {
            cuda_error = hipMemcpy((void *)h_vcnt, (void *)d_vcnt, sizeof(float)*ENV_DIM_X*ENV_DIM_Y, hipMemcpyDeviceToHost);
            if (cuda_error != hipSuccess){
                cout << "CUDA error in hipMemcpy: " << hipGetErrorString(cuda_error) << endl;
                exit(-1);
            }  
            write_vehicle_cnt_info(i, h_vcnt);
        }
    }
    hipDeviceSynchronize();
    cuda_error = hipMemcpy((void *)h_vcnt, (void *)d_vcnt, sizeof(float)*ENV_DIM_X*ENV_DIM_Y, hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess){
        cout << "CUDA error in hipMemcpy: " << hipGetErrorString(cuda_error) << endl;
        exit(-1);
    }  
    write_vehicle_cnt_info(N_ITER, h_vcnt);
    
    delete h_vcnt;
    delete h_vcap;
    delete h_turn;
}
